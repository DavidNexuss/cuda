#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stb/stb_image_write.h>
#include "util/buffer.h"
#include "util/debug.h"
#include "trace.h"

//SCENE
typedef struct {
  int   maxObjects;
  int   maxMaterials;
  int   maxMeshes;
  int   frameBufferWidth;
  int   frameBufferHeight;
  int   numThreads;
  int   iterationsPerThread;
  int   rayDepth;
  int   framesInFlight;
  float frameDelta;

} SceneDesc;

typedef struct _Scene {
  //Input buffer objects
  Buffer meshes;
  Buffer objects;
  Buffer materials;

  //Output buffer objects
  Buffer framebuffer;

  //Push constants
  Camera camera;
  int    objectCount   = -1;
  int    materialCount = -1;
  int    meshCount     = -1;

  //Scene configuration
  SceneDesc desc;

} Scene;

typedef struct {
  Object*   objects;
  Material* materials;
  Mesh*     meshes;
} SceneInput;

Scene sceneCreate(SceneDesc desc) {
  Scene scene;
  scene.desc        = desc;
  scene.meshes      = bufferCreate(sizeof(Mesh) * desc.maxMeshes);
  scene.objects     = bufferCreate(sizeof(Object) * desc.maxObjects);
  scene.materials   = bufferCreate(sizeof(Material) * desc.maxMaterials);
  scene.framebuffer = bufferCreate(3 * sizeof(float) * desc.frameBufferWidth * desc.frameBufferHeight * desc.framesInFlight);
  return scene;
}

SceneInput sceneInputHost(Scene* scene) {
  scene->objectCount = 0;
  return {
    (Object*)scene->objects.H,
    (Material*)scene->materials.H,
    (Mesh*)scene->meshes.H};
}

void sceneDestroy(Scene* scene) {
  bufferDestroy(&scene->meshes);
  bufferDestroy(&scene->objects);
  bufferDestroy(&scene->materials);
  bufferDestroy(&scene->framebuffer);
}

void sceneUpload(Scene* scene) {
  bufferUpload(&scene->materials, scene->materialCount * sizeof(Material));
  bufferUpload(&scene->meshes, scene->meshCount * sizeof(Mesh));
}

void sceneUploadObjects(Scene* scene) {
  bufferUpload(&scene->objects, scene->objectCount * sizeof(Object) * scene->desc.framesInFlight);
}

void sceneDownload(Scene* scene) {
  bufferDownload(&scene->framebuffer);
}

float* sceneGetFrame(Scene* scene, int index) {
  return (float*)&scene->framebuffer.H[index * 3 * sizeof(float) * scene->desc.frameBufferWidth * scene->desc.frameBufferHeight];
}

void sceneWriteFrame(Scene* scene, const char* path, int index) {
  stbi_write_hdr(path, scene->desc.frameBufferWidth, scene->desc.frameBufferHeight, 3, sceneGetFrame(scene, index));
}

//Execute path tracing on the scene with the given parameters
__global__ void pathTracingKernel(SceneInput sceneInput, Camera cam, int objectCount, int width, int height, float* fbo_mat, int iterationsPerThread, int maxDepth) {
  float u = blockIdx.x / float(width);
  float v = blockIdx.y / float(height);

  int pixelIdx = (blockIdx.x * width + blockIdx.y) * 3;
  int thread   = threadIdx.x;

  extern __shared__ float3 result[];

  float* fbo = fbo_mat + blockIdx.z * width * height * 3;

  float3  sro     = cam.origin;
  float3  srd     = make_float3(u * 2 - 1, v * 2 - 1, 1);
  Object* objects = sceneInput.objects + blockIdx.z * objectCount;


  //Perform path tracing using rd and ro
  /*
  float3 threadResult;
  for (int i = 0; i < iterationsPerThread; i++) {
    float3 partialResult = make_float3(0, 0, 0);

    float3 ro = sro;
    float3 rd = srd;

    for (int d = 0; d < maxDepth; d++) {
    }
  } */

  //Default uv gradient test
  fbo[pixelIdx]     = u;
  fbo[pixelIdx + 1] = v;
  fbo[pixelIdx + 2] = threadIdx.x / float(blockDim.x);
}

void sceneRun(Scene* scene) {
  dim3 numBlocks           = dim3(scene->desc.frameBufferWidth, scene->desc.frameBufferHeight, scene->desc.framesInFlight);
  int  numThreads          = scene->desc.numThreads;
  int  iterationsPerThread = scene->desc.iterationsPerThread;
  LOG("Running path tracing kernel [%d, %d, %d] with %d threads, iterations per thread: %d\n", numBlocks.x, numBlocks.y, numBlocks.z, numThreads, iterationsPerThread);

  pathTracingKernel<<<numBlocks, numThreads, sizeof(float) * 3 * numThreads>>>({(Object*)scene->objects.D, (Material*)scene->materials.H, (Mesh*)scene->meshes.H},
                                                                               scene->camera, scene->objectCount, scene->desc.frameBufferWidth, scene->desc.frameBufferHeight, (float*)scene->framebuffer.D, iterationsPerThread, scene->desc.rayDepth);
}

void defaultScene(Scene* scene);
int  defaultSceneLoop(Object* objects, float t);

void programRun(const char* path, int width, int height, void(initScene)(Scene*), int(initSceneFrame)(Object*, float t)) {

  SceneDesc sceneDesc           = {};
  sceneDesc.maxMeshes           = 300;
  sceneDesc.maxObjects          = 400;
  sceneDesc.maxMaterials        = 300;
  sceneDesc.frameBufferWidth    = width;
  sceneDesc.frameBufferHeight   = height;
  sceneDesc.numThreads          = 4;
  sceneDesc.iterationsPerThread = 4;
  sceneDesc.rayDepth            = 4;
  sceneDesc.framesInFlight      = 8;
  sceneDesc.frameDelta          = 0.1;

  Scene scene = sceneCreate(sceneDesc);

  //Inits scene materials and meshes
  {
    initScene(&scene);
    sceneUpload(&scene);
  }

  //Inits scene objects
  {
    float   t   = 0;
    Object* src = (Object*)scene.objects.H;

    for (int i = 0; i < sceneDesc.framesInFlight; i++) {
      int objects = initSceneFrame((Object*)scene.objects.H, t);
      t += sceneDesc.frameDelta;
      scene.objectCount = objects;
      src += objects;
    }

    sceneUploadObjects(&scene);
  }

  sceneRun(&scene);
  sceneDownload(&scene);
  sceneWriteFrame(&scene, path, 0);
  sceneDestroy(&scene);
}

int main(int argc, char** argv) {

  programRun("result.hdr", 1024, 1024, defaultScene, defaultSceneLoop);
  programRun("result2.hdr", 1024 * 2, 1024 * 2, defaultScene, defaultSceneLoop);

  LOG("[STATS] Peak memory use: %d\n", gBufferPeakAllocatedSize);
  LOG("[STATS] Memory leak : %d\n", gBufferTotalAllocatedSize);
}

void defaultScene(Scene* scene) {
  SceneInput inp = sceneInputHost(scene);

  int meshIdx     = 0;
  int materialIdx = 0;
  int objectIdx   = 0;

  inp.meshes[meshIdx++] = meshPlain(make_float3(0, 1, 0));
  inp.meshes[meshIdx++] = meshPlain(make_float3(1, 0, 0));
  inp.meshes[meshIdx++] = meshPlain(make_float3(0, 0, 1));
  inp.meshes[meshIdx++] = meshPlain(make_float3(0, 1, 1));

  inp.materials[materialIdx++] = {
    .kd      = vec3(0.5, 0.7, 0.8),
    .ks      = vec3(0.2, 0.4, 0.5),
    .ka      = vec3(0.1, 0.1, 0.1),
    .fresnel = 0.1,
    .ior     = 1.01};

  inp.materials[materialIdx++] = {
    .kd      = vec3(0.8, 0.7, 0.2),
    .ks      = vec3(0.2, 0.2, 0.2),
    .ka      = vec3(0.1, 0.1, 0.1),
    .fresnel = 0.0,
    .ior     = 1.01};

  inp.materials[materialIdx++] = {
    .kd      = vec3(10.8, 10.7, 10.2),
    .ks      = vec3(0.2, 0.2, 0.2),
    .ka      = vec3(0.1, 0.1, 0.1),
    .fresnel = 0.0,
    .ior     = 1.01};

  inp.materials[materialIdx++] = {
    .kd      = vec3(0.01, 0.1, 0.2),
    .ks      = vec3(0.8, 1.0, 1.0),
    .ka      = vec3(0.1, 0.1, 0.1),
    .fresnel = 1.0,
    .ior     = 1.01};

  inp.objects[objectIdx++] = {.material = 0, .mesh = 1, .origin = vec3(0, 1, 1)};
  inp.objects[objectIdx++] = {.material = 1, .mesh = 2, .origin = vec3(-1, 1, 1)};
  inp.objects[objectIdx++] = {.material = 2, .mesh = 3, .origin = vec3(-2, -1, 1)};
  inp.objects[objectIdx++] = {.material = 1, .mesh = 0, .origin = vec3(2, 1, 1)};
  inp.objects[objectIdx++] = {.material = 3, .mesh = 0, .origin = vec3(1, -1, 1)};

  scene->objectCount   = objectIdx;
  scene->materialCount = materialIdx;
  scene->meshCount     = meshIdx;
}

int defaultSceneLoop(Object* objects, float t) {
  int objectIdx        = 0;
  objects[objectIdx++] = {.material = 0, .mesh = 1, .origin = vec3(t, 1, 1)};
  objects[objectIdx++] = {.material = 1, .mesh = 2, .origin = vec3(-1, 1, 1)};
  objects[objectIdx++] = {.material = 2, .mesh = 3, .origin = vec3(-2, -1, 1)};
  objects[objectIdx++] = {.material = 1, .mesh = 0, .origin = vec3(2, 1, 1)};
  objects[objectIdx++] = {.material = 3, .mesh = 0, .origin = vec3(1, -1, 1)};
  return objectIdx;
}
