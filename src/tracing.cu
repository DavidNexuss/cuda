#include "hip/hip_runtime.h"
#include <stdio.h>

extern "C" {
#include "scene.h"
}

__device__ float3 prod(float3 a, float3 b) {
  return make_float3(a.x * b.x, a.y * b.y, a.z * b.z);
}
__device__ float3 prodScalar(float3 a, float t) {
  return prod(a, make_float3(t, t, t));
}
__device__ float3 sub(float3 a, float3 b) {
  return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
}
__device__ float3 sum(float3 a, float3 b) {
  return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}

//This are cuda versions of basic linear algebra functionality
__device__ float3 lReflect(float3 rd, float3 normal);
__device__ float3 lRefract(float3 rd, float3 normal, float ior);
__device__ float3 lNormalize(float3 v);
__device__ float  lLen2(float3 a);
__device__ float  lLen(float3 a);

//Returns origin + direction * distance
__device__ float3 lAdvance(float3 origin, float3 direction, float distance);

//Retuns a normalized random direction
__device__ float3 lRandomDirection();

//Retuns a normalized random direction in a hemisphere from normalvector
__device__ float3 lRrandomDirectionHemisphere(float3 normalvector);

//Returns sky color
__device__ float3 lClearColorBackground(float3 rd, float3 ground, float3 orizon, float3 sky) {
  float t = rd.y;
  return sub(prodScalar(sky, t), prodScalar(ground, -t));
}

//Signed distance field functions combined with direction optimisation whenever possible
__device__ int sdfHitSphere(float3 ro, float3 rd, float radius, float* delta, float3* normal);
__device__ int sdfHitPlane(float3 ro, float3 rd, float3 normal, float* delta, float* normalDir);

__device__ float3 sampleTexture(dim3* rgb, float2 uv);

__host__ __device__ float3 pathTracing(int width, int height, int iterationsPerThread, int maxDepth, SceneInput input, int x, int y, int frame) { 
  float u = x / float(width);
  float v = y / float(height);

  return make_float3(u, v, 1);
}

__global__ void pathTracingKernel(int width, int height, float* fbo_mat, int iterationsPerThread, int maxDepth, SceneInput input) {
  float3 result = pathTracing(width, height,iterationsPerThread, maxDepth, input, blockIdx.x, blockIdx.y, blockIdx.z);
  
  int pixelIdx = (blockIdx.x * width + blockIdx.y) * 3;
  float* fbo = &fbo_mat[blockIdx.z * width * height * 3];
  fbo[pixelIdx]     = result.x;
  fbo[pixelIdx + 1] = result.y;
  fbo[pixelIdx + 2] = result.z;

}

static int jobIdCounter = 0;
void _sceneRun(Scene* scene) { 
  dim3 numBlocks           = dim3(scene->desc.frameBufferWidth, scene->desc.frameBufferHeight, scene->desc.framesInFlight);
  dim3 numThreads          = dim3(scene->desc.numThreads, 1, 1);
  int  iterationsPerThread = scene->desc.iterationsPerThread;
  int jobId = jobIdCounter;
  dprintf(2, "[CUDA %d ] Running path tracing kernel [%d, %d, %d] with %d threads, iterations per thread: %d\n", jobId, numBlocks.x, numBlocks.y, numBlocks.z, numThreads.x, iterationsPerThread);
  pathTracingKernel<<<numBlocks, numThreads>>> (numBlocks.x,numBlocks.y, (float*)scene->framebuffer.D, iterationsPerThread, scene->desc.rayDepth, sceneInputDevice(scene));
  dprintf(2, "[CUDA %d ] done\n", jobId);
  jobIdCounter++;

}
extern "C" {
void sceneRun(Scene* scene) {
  _sceneRun(scene);
}


void sceneRunCPU(Scene *scene) { 
  
  int jobId = jobIdCounter;
  int  numThreads          = scene->desc.numThreads;
  int  iterationsPerThread = scene->desc.iterationsPerThread;
  dprintf(2, "[CPU %d ] Running path tracing kernel in CPU iterations %d x %d \n", jobId, iterationsPerThread, numThreads);

  SceneInput inp = sceneInputHost(scene);
  for(int i =0; i < scene->desc.framesInFlight; i++) {
    float* fbo = sceneGetFrame(scene, i);
    for(int x = 0; x < scene->desc.frameBufferWidth; x++) { 
      for(int y = 0; y < scene->desc.frameBufferHeight; y++) { 
        int pixelIdx = (x* scene->desc.frameBufferWidth + y) * 3;
        float3 result = pathTracing(scene->desc.frameBufferWidth, scene->desc.frameBufferHeight, numThreads * iterationsPerThread, scene->desc.rayDepth, inp, x, y, i);

        fbo[pixelIdx]     = result.x;
        fbo[pixelIdx + 1] = result.y;
        fbo[pixelIdx + 2] = result.z;
      }
    }
  }

  dprintf(2, "[CPU %d ] Done \n", jobId);
  jobIdCounter++;
}
}

//Perform path tracing using rd and ro

#if 0
  float3 threadResult;
  for (int i = 0; i < iterationsPerThread; i++) {
    float3 partialResult = make_float3(0, 0, 0);

    float3 ro = sro;
    float3 rd = srd;

    for (int d = 0; d < maxDepth; d++) {
    }
  }
#endif


#if 0
  fbo[pixelIdx]   = result.x;
  fbo[pixelIdx + 1] = result.y;
  fbo[pixelIdx + 2] = result.z;
#endif
  //Default uv gradient test
